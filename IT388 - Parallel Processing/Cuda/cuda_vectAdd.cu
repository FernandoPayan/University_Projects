/* FILE: cuda_vectAdd.cu
Compile and run in Expanse cluster
1) Load module:
	module load cuda
2) Compile
nvcc -o cuda_vectAdd cuda_vectAdd.cu

3) a) Submit job
	sbatch jobCuda.jb

Name: Fernando Payan
IT 388

TODO: Implement add vectors using cuda.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include<sys/time.h>
#include<math.h>

/* Function that adds two vectors */
__global__ void addVectors(double *x, double *y, double*z, int n){
    //global thread ID
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    //make sure is not out of bounds
    if (index < n){
        z[index] = x[index] + y[index];
    }
}

/*--------------------------------
 Function to Print Vectors
 --------------------------------*/
void printVectors(double *a, double *b, double*c, int n) {
    int k = 0;
    for(k=0;k<n;k++)
    printf("\n %f + %f  = %f",  a[k] , b[k], c[k]);
}

/*--------------------------------
 Function to measure run time
 --------------------------------*/
double getTime(){
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec + t.tv_usec/1000000.0;

}

/*------------------------------------------------------------------*/
int main(int argc, char* argv[]) {

    double start, elapsed;
    double* z; double* x; double* y;
    int n;
    int i;
    int blockSize = 256;
    int numBlocks = ceil((double)n / blockSize);

    hipMallocManaged(&x, n * sizeof(double));
    hipMallocManaged(&y, n * sizeof(double));
    hipMallocManaged(&z, n * sizeof(double));

    // generate vectors
    for (i=0;i<n;i++){
        x[i]=sin(i)*sin(i);
        y[i]=cos(i)*cos(i);
    }
    start = getTime();
    addVectors<<<numBlocks, blockSize>>>(x, y, z, n);
    hipDeviceSynchronize();

    elapsed = getTime() - start;
    printf("\nN=%d; Elapsed time:%f\n",n,elapsed);
    // Print vectors
    if (n<21){
        printVectors(x, y, z,  n);
    }
    // Check if vector z is correct, this should equal 1
    double sum = 0;
    for(i=0; i<n; i++){
        sum += z[i];
    }
    printf("final result: %f\n", sum/n);
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}  /* main */
